#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <caliper/cali.h>
#include <caliper/cali-manager.h>
//#include <adiak.hpp>

int THREADS;
int BLOCKS;
int NUM_VALS;

const char* bitonic_sort_step_region = "bitonic_sort_step";
const char* cudaMemcpy_host_to_device = "cudaMemcpy_host_to_device";
const char* cudaMemcpy_device_to_host = "cudaMemcpy_device_to_host";

float random_float() {
  return (float)rand() / (float)RAND_MAX;
}

void array_fill(float* arr, int length) {
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}

__global__ void quick_sort_step(float* dev_values, int l, int h) {
  unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
  
  if (i >= l && i <= h) {
    float piv = dev_values[l];
    int left = l;
    int right = h;
    
    while (left <= right) {
      while (left <= h && dev_values[left] <= piv) {
        left++;
      }
      while (right >= l && dev_values[right] > piv) {
        right--;
      }
      
      if (left <= right) {
        // Swap elements if necessary
        float temp = dev_values[left];
        dev_values[left] = dev_values[right];
        dev_values[right] = temp;
        left++;
        right--;
      }
    }
  }
}

/**
 * Inplace bitonic sort using CUDA.
 */
void quick_sort(float* values, int low, int high) {
  float* dev_values;
  size_t size = (high - low + 1) * sizeof(float);

  hipMalloc((void**)&dev_values, size);
  
  // MEM COPY FROM HOST TO DEVICE
  CALI_MARK_BEGIN(cudaMemcpy_host_to_device);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);
  CALI_MARK_END(cudaMemcpy_host_to_device);

  dim3 blocks(BLOCKS, 1);    /* Number of blocks   */
  dim3 threads(THREADS, 1);  /* Number of threads  */

  // Major step
  CALI_MARK_BEGIN(bitonic_sort_step_region);
  quick_sort_step<<<blocks, threads>>>(dev_values, 0, size - 1);
  hipDeviceSynchronize();
  CALI_MARK_END(bitonic_sort_step_region);

  // MEM COPY FROM DEVICE TO HOST
  CALI_MARK_BEGIN(cudaMemcpy_device_to_host);
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  CALI_MARK_END(cudaMemcpy_device_to_host);
  hipFree(dev_values);
}

int main(int argc, char* argv[]) {
  THREADS = atoi(argv[1]);
  NUM_VALS = atoi(argv[2]);
  BLOCKS = NUM_VALS / THREADS;

  printf("Number of threads: %d\n", THREADS);
  printf("Number of values: %d\n", NUM_VALS);
  printf("Number of blocks: %d\n", BLOCKS);

  cali::ConfigManager mgr;
  mgr.start();
  float* values = (float*)malloc(NUM_VALS * sizeof(float));

  clock_t start, stop;
  start = clock();
  quick_sort(values, 0, NUM_VALS - 1);
  stop = clock();

 // print_elapsed(start, stop);

  mgr.stop();
  mgr.flush();
}
